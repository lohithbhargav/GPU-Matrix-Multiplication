/*
 * _MATRIXMUL_GPU_CU_
 *
 * 2022 Mert SIDE
 *
 * CS5375 Computer Systems Organization and Architecture 
 * Guest Lecture: GPU Programming
 *
 * Multiplying two matrices on the GPU
 *
 */

 
#include <hip/hip_runtime.h>
#include <iostream>
 #include <stdio.h>
 #include <stdlib.h>
 
 // Kernel function to multiple the elements of two arrays
 __global__
 void GPUmatmul(int N, double *x, double *y, double *ans)
 {
  // Calculate the index for each thread row and column using blockDim
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  for(int k = 0; k < N; k++) {
    ans[index] += (x[blockIdx.x*blockDim.x+k] * y[k*blockDim.y+threadIdx.y]);
  }
 }
 
 // function to check whether we got correct value or not
 bool check(int N, double *ans)
 {
  for(int i = 0; i < N; i++) {
    for(int j = 0; j < N; j++) {
      if(ans[i*N+j] != 20.0) return false;
    }
   }
   return true;
 }
 
 int main(void)
 {
   // size of matrix
   int N = 1<<9; // binary left-shift: 1 * 2^9 = 512
   printf("Size of matrix (N) is %d by %d.\n", N, N);
   int iter = 3;
   clock_t t;
   
   // Martices
   double *x, *y, *ans;
 
   // Allocate Unified Memory - accessible from both CPU and GPU
   hipMallocManaged(&x, N*N*sizeof(double));
   hipMallocManaged(&y, N*N*sizeof(double));
   hipMallocManaged(&ans, N*N*sizeof(double));
   
   // initialize x,y and ans arrays on the host
   for (int i = 0; i < N; i++) {
     for(int j = 0; j < N; j++) {
       x[i*N+j] = 5;
       y[i*N+j] = (i==j?1:0);
       ans[i*N+j] = (double)0.000000000000;
     }
   }
 
   double avg=0;
   std::cout<<"Starting Optimized GPU computation"<<std::endl;
   // Run kernel on GPU
   for(int i = 0; i <= iter; i++) {
     t = clock();
     GPUmatmul<<<512,512>>>(N, x, y, ans); // we have updated the threads and blocks from v1
     hipDeviceSynchronize(); // Wait for GPU to finish before accessing on host
     t = clock() - t;
     if(i) avg += t; //we will ignore the first run
     // printf ("It took GPU-%d %f ms.\n",i,(((double)t)/CLOCKS_PER_SEC)*1000);
   }
   
   avg /= iter;
   avg /= CLOCKS_PER_SEC;
   avg *= 1000;
   printf("It took %lf ms on avg.\n", avg);
   if(check(N,ans)){
    std::cout<<"RUN OK."<<std::endl; // Check for errors
    }
   else{
    std::cout<<"RUN NOT OK."<<std::endl;
    }
 
   // Free memory
   hipFree(x);
   hipFree(y);
   hipFree(ans);
 
   return 0;
 }
 /* EOF */