/*
 * _MATRIXMUL_GPU_CU_
 *
 * 2022 Mert SIDE
 *
 * CS5375 Computer Systems Organization and Architecture 
 * Guest Lecture: GPU Programming
 *
 * Multiplying two matrices on the GPU
 *
 */

 
#include <hip/hip_runtime.h>
#include <iostream>
 #include <stdio.h>
 #include <stdlib.h>
 
 // Kernel function to multiple the elements of two arrays
 __global__
 void GPUmatmul(int N, double *x, double *y, double *ans)
 {
  int t = (blockDim.x*blockDim.y)*threadIdx.z+(threadIdx.y*blockDim.x)+(threadIdx.x); // thread number of a thread inside a particular block
  int b = (gridDim.x*gridDim.y)*blockIdx.z+(blockIdx.y*gridDim.x)+(blockIdx.x); // block number of a block inside the grid
  int T = blockDim.x*blockDim.y*blockDim.z; // total number of blocks
  int B = gridDim.x*gridDim.y*gridDim.z; // total number of threads per block
  for(int i = b; i < N; i+=B) {
    for(int j = t; j < N; j+=T) {
      for(int k = 0; k < N; k+=1) {
        ans[i*N+j] += (x[i*N+k] * y[k*N+j]);
      }
    }
  }
 }
 
 // function to check whether we got correct value or not
 bool check(int N, double *ans)
 {
  for(int i = 0; i < N; i++) {
    for(int j = 0; j < N; j++) {
       if(ans[i*N+j] != 20.0) return false;
     }
   }
   return true;
 }
 
 int main(void)
 {
   // size of matrix
   int N = 1<<9; // binary left-shift: 1 * 2^9 = 512
   printf("Size of matrix (N) is %d by %d.\n", N, N);
   int iter = 3;
   clock_t t;
   
   // Martices
   double *x, *y, *ans;
 
   // Allocate Unified Memory - accessible from both CPU and GPU
   hipMallocManaged(&x, N*N*sizeof(double));
   hipMallocManaged(&y, N*N*sizeof(double));
   hipMallocManaged(&ans, N*N*sizeof(double));
   
   // initialize x,y and ans arrays on the host
   for (int i = 0; i < N; i++) {
     for(int j = 0; j < N; j++) {
       x[i*N+j] = 5;
       y[i*N+j] = (i==j?1:0);
       ans[i*N+j] = (double)0.000000000000;
     }
   }
 
   // ..........................................................................
   double avg=0;
   std::cout<<"Starting Optimized GPU computation"<<std::endl;
   // Run kernel on GPU
   for(int i = 0; i <= iter; i++) {
     t = clock();
     GPUmatmul<<<dim3(16,4,4),dim3(8,8,8)>>>(N, x, y, ans); // we used dim3 to specify dimensions of vector type based on uint3
     hipDeviceSynchronize(); // Wait for GPU to finish before accessing on host
     t = clock() - t;
     if(i) avg += t; //we will ignore the first run
     // printf ("It took GPU-%d %f ms.\n",i,(((double)t)/CLOCKS_PER_SEC)*1000);
   }
   
   avg /= iter;
   avg /= CLOCKS_PER_SEC;
   avg *= 1000;
   printf("It took %lf ms on avg.\n", avg);
   if(check(N,ans)) std::cout<<"RUN OK."<<std::endl; // Check for errors
   else std::cout<<"RUN NOT OK."<<std::endl;
 
   // Free memory
   hipFree(x);
   hipFree(y);
   hipFree(ans);
 
   return 0;
 }
 /* EOF */